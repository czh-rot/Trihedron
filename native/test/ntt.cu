#include "hip/hip_runtime.h"
#include "ntt.cuh"

__global__ void ForwardCore(Data* polynomial_in, Data* polynomial_out, Root* root_of_unity_table,
                            Modulus* modulus, int shared_index, int logm,
                            int outer_iteration_count, int N_power,
                            bool zero_padding, bool not_last_kernel,
                            bool reduction_poly_check, int mod_count)
{
    const int idx_x = threadIdx.x; // 8
    const int idx_y = threadIdx.y; // 32
    const int block_x = blockIdx.x; // 64

    extern __shared__ Data shared_memory[];

    int t_2 = N_power - logm - 1; // logm=0, t_2 = 14
    location_t offset = 1 << (N_power - logm - 1); // 2^14, N_power=15
    int t_ = shared_index; // 8
    location_t m = (location_t)1 << logm; // m = 0

    location_t global_addresss =
        idx_x +
        (location_t)(idx_y * (offset / (1 << (outer_iteration_count - 1)))) + // o_i_c=6, the number of stages
        (location_t)(blockDim.x * block_x) +
        (location_t)(2 * 1 * offset);

    location_t omega_addresss =
        idx_x +
        (location_t)(idx_y * (offset / (1 << (outer_iteration_count - 1)))) +
        (location_t)(blockDim.x * block_x) + (location_t)(1 * offset);


    location_t shared_addresss = (idx_x + (idx_y * blockDim.x));

    // Load data from global & store to shared
    shared_memory[shared_addresss] = polynomial_in[global_addresss];
    shared_memory[shared_addresss + (blockDim.x * blockDim.y)] =
        polynomial_in[global_addresss + offset];

    int t = 1 << t_;
    int in_shared_address = ((shared_addresss >> t_) << t_) + shared_addresss;
    location_t current_root_index;
    if (not_last_kernel)
    {
#pragma unroll
        for (int lp = 0; lp < outer_iteration_count; lp++)
        {
            __syncthreads();
            if (reduction_poly_check)
            {  // X_N_minus
                current_root_index = (omega_addresss >> t_2) + (location_t)(mod_index << N_power);
            }
            else
            {  // X_N_plus
                current_root_index = m + (omega_addresss >> t_2) + (location_t)(mod_index << N_power);
            }
            CooleyTukeyUnit(shared_memory[in_shared_address],
                            shared_memory[in_shared_address + t],
                            root_of_unity_table[current_root_index], modulus[mod_index]);

            t = t >> 1;
            t_2 -= 1;
            t_ -= 1;
            m <<= 1;

            in_shared_address =
                ((shared_addresss >> t_) << t_) + shared_addresss;
            //__syncthreads();
        }
        __syncthreads();
    }
    else
    {
#pragma unroll
        for (int lp = 0; lp < (shared_index - 5); lp++) // 4 for 512 thread
        {
            __syncthreads();
            if (reduction_poly_check)
            {  // X_N_minus
                current_root_index = (omega_addresss >> t_2) + (location_t)(mod_index << N_power);
            }
            else
            {  // X_N_plus
                current_root_index = m + (omega_addresss >> t_2) + (location_t)(mod_index << N_power);
            }

            CooleyTukeyUnit(shared_memory[in_shared_address],
                            shared_memory[in_shared_address + t],
                            root_of_unity_table[current_root_index], modulus[mod_index]);

            t = t >> 1;
            t_2 -= 1;
            t_ -= 1;
            m <<= 1;

            in_shared_address =
                ((shared_addresss >> t_) << t_) + shared_addresss;
            //__syncthreads();
        }
        __syncthreads();

#pragma unroll
        for (int lp = 0; lp < 6; lp++)
        {
            if (reduction_poly_check)
            {  // X_N_minus
                current_root_index = (omega_addresss >> t_2) + (location_t)(mod_index << N_power);
            }
            else
            {  // X_N_plus
                current_root_index = m + (omega_addresss >> t_2) + (location_t)(mod_index << N_power);
            }
            CooleyTukeyUnit(shared_memory[in_shared_address],
                            shared_memory[in_shared_address + t],
                            root_of_unity_table[current_root_index], modulus[mod_index]);

            t = t >> 1;
            t_2 -= 1;
            t_ -= 1;
            m <<= 1;

            in_shared_address =
                ((shared_addresss >> t_) << t_) + shared_addresss;
        }
        __syncthreads();
    }

    polynomial_out[global_addresss] = shared_memory[shared_addresss];
    polynomial_out[global_addresss + offset] =
        shared_memory[shared_addresss + (blockDim.x * blockDim.y)];
}