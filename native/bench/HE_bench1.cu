#include <cstdlib>  // For atoi or atof functions
#include <random>

#include "../src/keygeneration.cuh"
#include "../src/encoder.cuh"
#include "../src/encryptor.cuh"
#include "../src/decryptor.cuh"
#include "../src/operator.cuh"

#include <fstream>

#define DEFAULT_MODULUS

using namespace std;


/*
// tests

cmake . -D CMAKE_CUDA_ARCHITECTURES=86 -D MODULAR_REDUCTION_TYPE=0 -B./cmake-build 

cmake --build ./cmake-build/ --target HE_bench1 --parallel
./cmake-build/HE_bench1 12 1
*/



int main(int N, int sizek, int sizev)
{
    Parameters contxt("BFV", 32768, PrimePool::security_level::HES_128);

    cout << "/ --------------------------------------------------------- /" << endl;
    cout << "|                 ~ WELCOME TO BFV GPU LIBRARY ~                 " << endl;
    cout << "| Encryption Parameters :" << endl;
    cout << "|  - Scheme: " << contxt.scheme << endl;
    cout << "|  - Poly Modulus Degree: " << contxt.n << endl;
    cout << "|  - Coeff Modulus Size: " << contxt.total_bits << " bits & Coeff Modulus Count: " << contxt.coeff_modulus << endl;
    cout << "|  - Plain Modulus: " << contxt.plain_modulus_.value << endl;
    cout << "/ --------------------------------------------------------- /" << endl;

    int coeff_modulus = contxt.coeff_modulus;
    const int n = contxt.n;
    const int row_size = n / 2;

    Secretkey secret_key(contxt);
    Publickey public_key(contxt);
    Relinkey relin_key(contxt);
    Galoiskey galois_key(contxt);

    HESecretkeygen(secret_key, contxt);
    HEPublickeygen(public_key, secret_key, contxt);
    HERelinkeygen(relin_key, secret_key, contxt);
    HEGaloiskeygen(galois_key, secret_key, contxt);

    Data* message = (Data*)malloc(sizeof(Data) * n);
    for (int i = 0; i < n; i++) {

        message[i] = 8;
    }
    
    message[0] = 1;
    message[1] = 12;
    message[2] = 23;
    message[3] = 31;
    message[row_size] = 41;
    message[row_size + 1] = 54;
    message[row_size + 2] = 6;
    message[row_size + 3] = 100; // message is the massage before plaintext

    Message M1(message, contxt);
    Plaintext P1(contxt);
    Plaintext P3(contxt);

    const int test_count = 50;
    float time_encoding_measurements[test_count];
    float time_decoding_measurements[test_count];
    float time_encryption_measurements[test_count];
    float time_decryption_measurements[test_count];
    float time_addition_measurements[test_count];
    float time_multiplication_measurements[test_count];
    float time_relinearization_measurements[test_count];
    float time_rotation_measurements[test_count];
    float time_plain_multiplication_measurements[test_count];
    for (int loop = 0; loop < test_count; loop++)
    {
        float time = 0;
        hipEvent_t start_encoding, stop_encoding;
        hipEventCreate(&start_encoding); hipEventCreate(&stop_encoding);
    
        HEEncoder encoder(contxt);
        hipEventRecord(start_encoding);
        // Encode 
        encoder.encode(P1, M1);
        
        hipEventRecord(stop_encoding);
        hipEventSynchronize(stop_encoding);
        hipEventElapsedTime(&time, start_encoding, stop_encoding);
        time_encoding_measurements[loop] = time;
        
        ///////////////////////////////////////////////////////////////////////////////
    
        Ciphertext C1(contxt);

        hipEvent_t start_encryption, stop_encryption;
        hipEventCreate(&start_encryption); hipEventCreate(&stop_encryption);

        HEEncryptor encryptor(contxt, public_key);

        hipEventRecord(start_encryption);
        // Encrypt
        encryptor.encrypt(C1, P1);

        hipEventRecord(stop_encryption);
        hipEventSynchronize(stop_encryption);
        hipEventElapsedTime(&time, start_encryption, stop_encryption);
        time_encryption_measurements[loop] = time;

        ///////////////////////////////////////////////////////////////////////////////

        Ciphertext C_mul(contxt);

        hipEvent_t start_multiplication, stop_multiplication;
        hipEventCreate(&start_multiplication); hipEventCreate(&stop_multiplication);

        HEOperator operators(contxt);

        hipEventRecord(start_multiplication);

        operators.multiply(C1, C1, C_mul);

        hipEventRecord(stop_multiplication);
        hipEventSynchronize(stop_multiplication);
        hipEventElapsedTime(&time, start_multiplication, stop_multiplication);
        time_multiplication_measurements[loop] = time;

        ///////////////////////////////////////////////////////////////////////////////

        hipEvent_t start_relinearization, stop_relinearization;
        hipEventCreate(&start_relinearization); hipEventCreate(&stop_relinearization);

        hipEventRecord(start_relinearization);

        operators.relinearize_inplace(C_mul, relin_key);

        hipEventRecord(stop_relinearization);
        hipEventSynchronize(stop_relinearization);
        hipEventElapsedTime(&time, start_relinearization, stop_relinearization);
        time_relinearization_measurements[loop] = time;

        ///////////////////////////////////////////////////////////////////////////////

        Ciphertext C2(contxt);

        hipEvent_t start_rotation, stop_rotation;
        hipEventCreate(&start_rotation); hipEventCreate(&stop_rotation);

        hipEventRecord(start_rotation);

        operators.rotate(C_mul, C2, galois_key, 1);

        hipEventRecord(stop_rotation);
        hipEventSynchronize(stop_rotation);
        hipEventElapsedTime(&time, start_rotation, stop_rotation);
        time_rotation_measurements[loop] = time;

        ///////////////////////////////////////////////////////////////////////////////

        hipEvent_t start_addition, stop_addition;
        hipEventCreate(&start_addition); hipEventCreate(&stop_addition);

        hipEventRecord(start_addition);

        operators.add_inplace(C2, C2);
        operators.HESubstractionInplace();

        hipEventRecord(stop_addition);
        hipEventSynchronize(stop_addition);
        hipEventElapsedTime(&time, start_addition, stop_addition);
        time_addition_measurements[loop] = time;

        ///////////////////////////////////////////////////////////////////////////////

        hipEvent_t start_plain_multiplication, stop_plain_multiplication;
        hipEventCreate(&start_plain_multiplication); hipEventCreate(&stop_plain_multiplication);

        hipEventRecord(start_plain_multiplication);

        operators.multiply_plain(C2, P3, C2);

        hipEventRecord(stop_plain_multiplication);
        hipEventSynchronize(stop_plain_multiplication);
        hipEventElapsedTime(&time, start_plain_multiplication, stop_plain_multiplication);
        time_plain_multiplication_measurements[loop] = time;

        ///////////////////////////////////////////////////////////////////////////////

        Message M2(contxt);
        Plaintext P2(contxt);
        
        HEDecryptor decryptor(contxt, secret_key);

        hipEvent_t start_decryption, stop_decryption;
        hipEventCreate(&start_decryption); hipEventCreate(&stop_decryption);

        hipEventRecord(start_decryption);

        decryptor.decrypt(P2, C2);

        hipEventRecord(stop_decryption);
        hipEventSynchronize(stop_decryption);
        hipEventElapsedTime(&time, start_decryption, stop_decryption);
        time_decryption_measurements[loop] = time;

        ///////////////////////////////////////////////////////////////////////////////

        hipEvent_t start_decoding, stop_decoding;
        hipEventCreate(&start_decoding); hipEventCreate(&stop_decoding);

        hipEventRecord(start_decoding);

        encoder.decode(M2, P2);

        hipEventRecord(stop_decoding);
        hipEventSynchronize(stop_decoding);
        hipEventElapsedTime(&time, start_decoding, stop_decoding);
        time_decoding_measurements[loop] = time;
        

    }

    float sum_encoding = 0.0;
    float sum_decoding = 0.0;
    float sum_encryption = 0.0;
    float sum_decryption = 0.0;
    float sum_addition = 0.0;
    float sum_multiplication = 0.0;
    float sum_relinearization = 0.0;
    float sum_rotation = 0.0;
    float sum_plain_multiplication = 0.0;

    for (int i = 0; i < test_count; ++i)
    {
        sum_encoding += time_encoding_measurements[i];
        sum_decoding += time_decoding_measurements[i];
        sum_encryption += time_encryption_measurements[i];
        sum_decryption += time_decryption_measurements[i];
        sum_addition += time_addition_measurements[i];
        sum_multiplication += time_multiplication_measurements[i];
        sum_relinearization += time_relinearization_measurements[i];
        sum_rotation += time_rotation_measurements[i];
        sum_plain_multiplication += time_plain_multiplication_measurements[i];
    }

    cout << endl << "/ ----------------------- Timings ------------------------- /" << endl;
    cout << "Encoding timing:             " << sum_encoding / test_count << " ms"<< endl;
    cout << "Decoding timing:             " << sum_decoding / test_count << " ms"<< endl;
    cout << "Encryption timing:           " << sum_encryption / test_count << " ms"<< endl;
    cout << "Decryption timing:           " << sum_decryption / test_count << " ms"<< endl;
    cout << "Addition timing:             " << sum_addition / test_count << " ms"<< endl;
    cout << "Multiplication timing:       " << sum_multiplication / test_count << " ms"<< endl;
    cout << "Relinearization timing:      " << sum_relinearization / test_count << " ms"<< endl;
    cout << "Rotation timing:             " << sum_rotation / test_count << " ms"<< endl;
    cout << "Plain_Multiplication timing: " << sum_plain_multiplication / test_count << " ms"<< endl;
    cout << "/ --------------------------------------------------------- /" << endl;

    // int batch = 
    // int sizek = atoi(argv[1]);
    // int sizev = atoi(argv[2]);
    // if (batch != 1 || sizek != 32|| sizev != 32) {
    //     cout << "Please Input 1, 32, 32 as the parameters!" << endl;
    //     return EXIT_SUCCESS;
    // }

    int test_count2 = 10;
    float time_QE_measurements[test_count2];
    float time_EC_measurements[test_count2];
    float time_PIR_measurements[test_count2];
    HEEncryptor encryptor(contxt, public_key);
    HEEncoder encoder(contxt);
    HEOperator operators(contxt);
    float time = 0;
    for (int ii = 0; ii < test_count2; ii++) {
        Data* m1 = (Data*)malloc(sizeof(Data) * n);
        Data* m2 = (Data*)malloc(sizeof(Data) * n);
        for (int i = 0; i < n; i++) {
            if (i < n/2) {
                m1[i] = 8;
            } else {
                m2[i] = 1;
            } 
        }
        Message Mask1(m1, contxt);
        Message Mask2(m1, contxt);
        Plaintext PMask1(contxt);
        Plaintext PMask2(contxt);
        Ciphertext t1(contxt);
        Ciphertext t2(contxt);
        Ciphertext C1(contxt);
        Ciphertext C2(contxt);
        hipEvent_t start_QE, stop_QE;
        hipEventCreate(&start_QE); hipEventCreate(&stop_QE);
        hipEventRecord(start_QE);
    }

    QueryExpansion(operators, C1, 10);
    

    return EXIT_SUCCESS;
}



vector<Ciphertext> QueryExpansion(HEOperator operator, Ciphertext qc, vector<Plaintext> mask, Galoiskey galois_key) {
    int lk = mask.size();
    vector<Ciphertext> Q(lk);
    Ciphertext tmp;
    for (int i = 0; i < lk; ++i) {
        operator.multiply_plain(qc, mask[i], Q[i]);
        for (int j = 0; j < 14; ++j) {
            operator.rotate(Q[i], tmp, galois_key, 1);
            operator.addinplace(Q[i], tmp);
        }
    }
    return Q;
}

// Process a group in a serial manner
vector<Ciphertext> EqualityCheck(HEOperator operator, vector<Ciphertext> Q, vector<vecotr<Ciphertext>> group_ik, Ciphertext c, Ciphertext l, Relinkey relin_key) {
    int kli = group_ik[0].size() - 1;
    int num = group_ik.size();
    vector<Ciphertext> I(num);
    for (int i = 0; i < num; ++i) {
        Ciphertext tmp(kli);
        for (int j = 0; j < kli; ++j) {
            operator.sub(Q[j], group_ik[i][j], tmp[j]);
            for (int k = 0; k < 16; ++k) { // T = 2^16+1
                operator.multiply(tmp[j], tmp[j], tmp[j]);
                operator.relinearize_inplace(tmp[j], relin_key);
            }
            operator.sub(c, tmp[j], tmp[j]);
            operator.addinplace(I[i], tmp[j]);
        }
        Ciphertext f;
        oprator.sub(Q[Q.size()-1], group_ik[i][kli], f);
        for (int k = 0; k < 16; ++k) { // T = 2^16+1
                operator.multiply(f, f, f);
                operator.relinearize_inplace(f, relin_key);
            }
        operator.sub(c, f, f);
        operator.addinplace(I[i], f);
        // todo Bootstrap
        operator.sub(I[i], l, I[i]);
        for (int k = 0; k < 16; ++k) { // T = 2^16+1
            operator.multiply(I[i], I[i], I[i]);
            operator.relinearize_inplaceI[i], relin_key);
        }
        operator.sub(c, I[i], I[i]);
    }
    return I;
}

// Process a group in a serial manner
Ciphertext IndexPIR(HEOperator operator, vector<Ciphertext> I, vector<vecotr<Ciphertext>> group_iv, Galoiskey galois_key) {
    int vli = group_iv[0].size();
    int num = group_iv.size();
    Ciphertext Ans;
    vector<Ciphertext> v;
    Ciphertext tmp;
    for (int i = 0; i < num; ++i) {
        for (int j = 0; j < vli; ++j) {
            operator.multiiply_plain(I[i], group_iv[i][j], tmp);
            operator.addinplace(v[j], tmp);
        }
    }
    for (int jj = 1; jj < vli; ++jj) {
        operator.rotate(v[jj], tmp, galois_key, 1);
        operator.addinplace(Ans, tmp);
    }
    operator.addinplace(Ans, v[0]);
    return Ans;
}


